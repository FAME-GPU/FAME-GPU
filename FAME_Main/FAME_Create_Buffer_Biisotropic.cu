#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include <assert.h>

int FAME_Create_Buffer_Biisotropic(	CULIB_HANDLES*  cuHandles, 
						FFT_BUFFER* 	fft_buffer,
						LANCZOS_BUFFER* lBuffer,
						int N, int Nstep)
{
    int N3 = N * 3;
	size_t memsize;

    memsize = N3 * sizeof(hipfftDoubleComplex);
    checkCudaErrors(hipMalloc((void**)&fft_buffer->d_A, 2*memsize));
    checkCudaErrors(hipMalloc((void**)&fft_buffer->dvec_x, memsize));
    checkCudaErrors(hipMalloc((void**)&fft_buffer->tmp, memsize));

    memsize = N3 * sizeof(hipDoubleComplex);
    checkCudaErrors(hipMalloc((void**)&cuHandles->N3_temp1, memsize));
    checkCudaErrors(hipMalloc((void**)&cuHandles->N3_temp2, memsize));

    memsize = Nstep * Nstep * sizeof(hipDoubleComplex);
    checkCudaErrors(hipMalloc((void**) &lBuffer->dz, memsize));

	memsize = Nstep * Nstep * sizeof(cmpx);
	lBuffer->z   = (cmpx*) malloc(memsize);   assert(lBuffer->z != NULL);

	memsize = Nstep * sizeof(double);
    lBuffer->T0  = (double*) malloc(memsize); assert(lBuffer->T0 != NULL);
    lBuffer->LT0 = (double*) malloc(memsize); assert(lBuffer->LT0 != NULL);
    lBuffer->T2  = (double*) malloc(memsize); assert(lBuffer->T2 != NULL);
    lBuffer->LT1 = (double*) malloc(memsize); assert(lBuffer->LT1 != NULL);

    memsize = (Nstep+1) * sizeof(double);
    lBuffer->T1  = (double*) malloc(memsize); assert(lBuffer->T1 != NULL); 

	return 0;
}

