#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "FAME_FFT_CUDA.cuh"
#include "FAME_Create_cublas.cuh"
#include "FAME_Create_Buffer.cuh"
#include "FAME_Create_Buffer_Biisotropic.cuh"
#include "FAME_Matrix_B_Isotropic.cuh"
#include "FAME_Malloc_mtx_C.h"
#include "FAME_Matrix_Lambdas.cuh"
#include "FAME_Matrix_Curl.h"
#include "FAME_Create_Frequency_txt.h"
#include "FAME_Save_Eigenvector.h"
#include "FAME_Profile.h"
#include "FAME_Destroy_Main.cuh"
#include "FAME_Fast_Algorithms_Isotropic.cuh"
#include "FAME_Fast_Algorithms_Biisotropic.cuh"
#include "FAME_Matrix_B_Biisotropic.cuh"
#include "FAME_Matrix_Vector_Production_Qrs.cuh"
#include "FAME_Matrix_Vector_Production_Pr.cuh"
#include "FAME_Print_Parameter.h"
#include "FAME_Create_Lambdas_txt.h"
#include <complex.h>

void FAME_Fast_Algorithms_Driver(
	double*        Freq_array,
	cmpx*          Ele_field_mtx,
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	MATERIAL 	   material,
	ES             es,
	LS             ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* lattice_type, PROFILE* Profile);

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type);
void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, PROFILE* Profile);
void Check_Residual(double* Freq_array, cmpx* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant);
void Check_Residual_Biiso(double* Freq_array, cmpx* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant);

int FAME_Main_Code(PAR Par, PROFILE* Profile)
{
	int Nx = Par.mesh.grid_nums[0];
    int Ny = Par.mesh.grid_nums[1];
	int Nz = Par.mesh.grid_nums[2];
	int Nd;
	int N  = Nx * Ny * Nz;
	int Ele_field_mtx_N=0;
	int Nwant = Par.es.nwant;
	int Nstep = Par.es.nstep;
	int N_wave_vec = Par.recip_lattice.Wave_vec_num;
	double wave_vec_array[3];

	double accum;
	struct timespec start, end;


	hipSetDevice(Par.flag.device);
	
    CULIB_HANDLES  cuHandles;
	FFT_BUFFER     fft_buffer;
	LANCZOS_BUFFER lBuffer;
	MTX_B          mtx_B;
	MTX_C          mtx_C;
	LAMBDAS        Lambdas;
    LAMBDAS_CUDA   Lambdas_cuda;

	FAME_Create_cublas(&cuHandles, Nx, Ny, Nz);

	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		Ele_field_mtx_N = N * 3;
	}
	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
	{
		Ele_field_mtx_N = N * 6;
	}
	double* Freq_array    = (double*) calloc(N_wave_vec * Nwant, sizeof(double));
	cmpx*   Ele_field_mtx = (cmpx*)   calloc(Ele_field_mtx_N * Nwant, sizeof(cmpx));
	
	
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		FAME_Create_Buffer(&cuHandles, &fft_buffer, &lBuffer, N, Nstep);
	}
	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
	{
		FAME_Create_Buffer_Biisotropic(&cuHandles, &fft_buffer, &lBuffer,N, Nstep);
	}

	
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		printf("= = = = FAME_Matrix_B_Isotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		size_t size = 3*N*sizeof(double);
		checkCudaErrors(hipMalloc((void**) &mtx_B.B_eps,    size));
		checkCudaErrors(hipMalloc((void**) &mtx_B.invB_eps, size));
		FAME_Matrix_B_Isotropic(mtx_B.B_eps, mtx_B.invB_eps, Par.material, N);
	}

	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
    {
		printf("= = = = FAME_Matrix_B_Biisotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		size_t  size = 3*N*sizeof(double);
		hipMalloc((void**) &mtx_B.B_eps, size);
		hipMalloc((void**) &mtx_B.B_mu, size);
		hipMalloc((void**) &mtx_B.invPhi,  size);

		size = 3*N*sizeof(hipDoubleComplex);
		hipMalloc((void**) &mtx_B.B_zeta, size);
		hipMalloc((void**) &mtx_B.B_zeta_s, size);
		hipMalloc((void**) &mtx_B.B_xi, size);

		FAME_Matrix_B_Biisotropic(N, Par.material, mtx_B.B_eps, mtx_B.B_mu, mtx_B.B_xi, mtx_B.B_zeta, mtx_B.B_zeta_s, mtx_B.invPhi );

	}


    FAME_Malloc_mtx_C(&mtx_C, N);
	FAME_Print_Parameter(Par );
	for(int i = 0; i < N_wave_vec; i++)
    //for(int i = 0; i < 1; i++)
	{
		Profile->idx = i;

		wave_vec_array[0] = Par.recip_lattice.WaveVector[3 * i];
    	wave_vec_array[1] = Par.recip_lattice.WaveVector[3 * i + 1];
    	wave_vec_array[2] = Par.recip_lattice.WaveVector[3 * i + 2];

    	printf("\033[40;33m= = Start to compute (%3d/%3d) WaveVector = [ % .6f % .6f % .6f ] = =\033[0m\n", i + 1, Par.recip_lattice.Wave_vec_num, wave_vec_array[0], wave_vec_array[1], wave_vec_array[2]);

		printf("= = = = FAME_Matrix_Curl  = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		FAME_Matrix_Curl(&mtx_C, wave_vec_array, Par.mesh.grid_nums, Par.mesh.edge_len, Par.mesh.mesh_len, Par.lattice);

		printf("= = = = FAME_Matrix_Lambdas = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		Nd = FAME_Matrix_Lambdas(&Lambdas_cuda, wave_vec_array, Par.mesh.grid_nums, Par.mesh.mesh_len, Par.lattice.lattice_vec_a, &Par, &Lambdas);
		

		printf("= = = = Check_Eigendecomp = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime(CLOCK_REALTIME, &start);
		Check_Eigendecomp(mtx_C, Lambdas, Lambdas_cuda, fft_buffer, cuHandles, Nx, Ny, Nz, Nd, N, Par.lattice.lattice_type, Profile);
		clock_gettime(CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);

		printf("= = = = FAME_Fast_Algorithms = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);
		FAME_Fast_Algorithms_Driver(Freq_array+i*Nwant, Ele_field_mtx, 
			cuHandles, lBuffer, fft_buffer, Lambdas_cuda, mtx_B, Par.material, Par.es, Par.ls,
			Nx, Ny, Nz, Nd, N, Par.lattice.lattice_type, Profile);
		clock_gettime (CLOCK_REALTIME, &end);
		Profile->es_time[Profile->idx] = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;
		
		printf("= = = = Check_Residual  = = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);
		if(strcmp(Par.material.material_type, "isotropic") == 0)
		{
			Check_Residual(Freq_array+i*Nwant, Ele_field_mtx, mtx_B, mtx_C, N, Nwant);
		}
		else if(strcmp(Par.material.material_type, "biisotropic") == 0)
		{
			Check_Residual_Biiso(Freq_array+i*Nwant, Ele_field_mtx, mtx_B, mtx_C, N, Nwant);
		}
		//getchar();
		clock_gettime (CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);

		if(Par.flag.save_eigen_vector)
		{
			printf("= = = = Save Eigen Vector = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
			FAME_Save_Eigenvector(Ele_field_mtx, Nwant, Ele_field_mtx_N, i);
		}

		Destroy_Lambdas(Lambdas, Lambdas_cuda, Par.lattice.lattice_type);

		FAME_Print_Profile(*Profile);

	}

	FAME_Create_Frequency_txt(Freq_array, Nwant, Profile->idx);
	
	FAME_Destroy_Main(cuHandles, fft_buffer, lBuffer, mtx_B, mtx_C, Freq_array, Ele_field_mtx);
	
	return 0;
}

void FAME_Fast_Algorithms_Driver(
	double*        Freq_array,
	cmpx*          Ele_field_mtx,
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	MATERIAL 	   material,
	ES             es,
	LS             ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* lattice_type, PROFILE* Profile)
{

	if(strcmp(material.material_type, "isotropic") == 0)
	{
		if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
		{

			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								  Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "Simple", Profile);
		}
		else
		{
			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								 Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "General", Profile);
		}
	}
	else if (strcmp(material.material_type, "biisotropic") == 0)
	{
		if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
			(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
			(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
		{		
			FAME_Fast_Algorithms_Biisotropic(Freq_array, Ele_field_mtx, cuHandles, Lambdas_cuda, lBuffer,fft_buffer, mtx_B,material ,Nx, Ny, Nz, Nd,
												es, ls, "Simple", Profile);		
		}
		else
		{
			FAME_Fast_Algorithms_Biisotropic(Freq_array, Ele_field_mtx, cuHandles,Lambdas_cuda, lBuffer, fft_buffer, mtx_B,material ,Nx, Ny, Nz, Nd,
				es, ls , "General",Profile);			
		}
	}
}

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type)
{
	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
        free(Lambdas.D_k);
        free(Lambdas.D_ks);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_k);
		hipFree(Lambdas_cuda.dD_ks);
	}

	else
	{
        free(Lambdas.D_kx);
        free(Lambdas.D_ky);
        free(Lambdas.D_kz);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_kx);
    	hipFree(Lambdas_cuda.dD_ky);
    	hipFree(Lambdas_cuda.dD_kz);
	}

    free(Lambdas.Lambda_q_sqrt);
    free(Lambdas.Pi_Qr);
    free(Lambdas.Pi_Pr);
    free(Lambdas.Pi_Qrs);
    free(Lambdas.Pi_Prs);

    hipFree(Lambdas_cuda.Lambda_q_sqrt);
	hipFree(Lambdas_cuda.dPi_Qr);
	hipFree(Lambdas_cuda.dPi_Pr);
	hipFree(Lambdas_cuda.dPi_Qrs);
	hipFree(Lambdas_cuda.dPi_Prs);
}

void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, PROFILE* Profile)
{
	int i;
	int N2 = N * 2;
	int Ele_field_mtx_N = N * 3;
	int N12 = N * 12;
	size_t size, dsizeEle_field_mtx_N, dsizeNd2;

	size = Ele_field_mtx_N * sizeof(cmpx);

	cmpx* vec_x    = (cmpx*) malloc(size);
	cmpx* vec_y    = (cmpx*) malloc(size);
	cmpx* vec_temp = (cmpx*) malloc(size);

	hipDoubleComplex* N3_temp1 = cuHandles.N3_temp1;
	hipDoubleComplex* N3_temp2 = cuHandles.N3_temp2;

	hipDoubleComplex* Nd2_temp;
	dsizeEle_field_mtx_N = Ele_field_mtx_N * sizeof(hipDoubleComplex);
	dsizeNd2 = Nd * 2 * sizeof(hipDoubleComplex);

	checkCudaErrors(hipMalloc((void**)&Nd2_temp, dsizeNd2));

	srand(time(NULL));

	for(i = 0; i < Ele_field_mtx_N; i++)
	//vec_x[i] = ((double) rand()/(RAND_MAX + 1.0))  for test
		vec_x[i] = ((double) rand()/(RAND_MAX + 1.0)) +  I * ((double) rand()/(RAND_MAX + 1.0));

	cmpx *vec_y_1, *vec_y_2, *vec_y_3;

	hipMemcpy(N3_temp1, vec_x, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
	{
		//FFT_CUDA(cuHandles, fft_buffer, N3_temp1, Nx, Ny, Nz, Lambdas_cuda.dD_ks, N3_temp2);
		FFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_ks, fft_buffer, cuHandles, Nx, Ny, Nz);
	}
	else
	{
		for(i = 0; i < 3; i++)
        	spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, -1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);
	vec_y_1 = &vec_y[0];  vec_y_2 = &vec_y[N];  vec_y_3 = &vec_y[N2];

	if(Nd == N - 1)
	{
		vec_y_1[0] = 0; vec_y_2[0] = 0; vec_y_3[0] = 0;
		for(i = 0; i < N - 1; i++)
		{
			vec_y_1[i + 1] = Lambdas.Lambda_x[i] * vec_y_1[i + 1];
			vec_y_2[i + 1] = Lambdas.Lambda_y[i] * vec_y_2[i + 1];
			vec_y_3[i + 1] = Lambdas.Lambda_z[i] * vec_y_3[i + 1];
		}
	}
	else
	{
		for(i = 0; i < N; i++)
		{
			vec_y_1[i] = Lambdas.Lambda_x[i] * vec_y_1[i];
			vec_y_2[i] = Lambdas.Lambda_y[i] * vec_y_2[i];
			vec_y_3[i] = Lambdas.Lambda_z[i] * vec_y_3[i];
		}
	}

	hipMemcpy(N3_temp1, vec_y, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		IFFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_k, fft_buffer, cuHandles, Nx, Ny, Nz);
		//IFFT_CUDA(cuHandles, fft_buffer, N3_temp1, Nx, Ny, Nz, Lambdas_cuda.dD_k, N3_temp2);
	}
	else
	{
		for(i = 0; i < 3; i++)
			spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, 1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);

	mtx_prod(&vec_temp[0] , mtx_C.C1_r, mtx_C.C1_c, mtx_C.C1_v, &vec_x[0] , N2, N);
	mtx_prod(&vec_temp[N] , mtx_C.C2_r, mtx_C.C2_c, mtx_C.C2_v, &vec_x[N] , N2, N);
	mtx_prod(&vec_temp[N2], mtx_C.C3_r, mtx_C.C3_c, mtx_C.C3_v, &vec_x[N2], N2, N);

	size = N * sizeof(cmpx);
	cmpx* test_x = (cmpx*) malloc(size);
	cmpx* test_y = (cmpx*) malloc(size);
	cmpx* test_z = (cmpx*) malloc(size);

	vec_plus(test_x, 1.0, &vec_temp[0] , -1.0, &vec_y[0] , N);
	vec_plus(test_y, 1.0, &vec_temp[N] , -1.0, &vec_y[N] , N);
	vec_plus(test_z, 1.0, &vec_temp[N2], -1.0, &vec_y[N2], N);
	
	double C1_error = vec_norm(test_x, N);
    double C2_error = vec_norm(test_y, N);
    double C3_error = vec_norm(test_z, N);

	free(test_x); free(test_y); free(test_z);

	cmpx* Qrs_x = (cmpx*) malloc(2*Nd*sizeof(cmpx));

	hipMemcpy(N3_temp1, vec_x, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Lambdas_cuda.dD_ks, Lambdas_cuda.dPi_Qrs, Nx, Ny, Nz, Nd, Profile);
	}
	else
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Qrs, Nx, Ny, Nz, Nd, Profile);
	}

	hipMemcpy(Qrs_x, Nd2_temp, dsizeNd2, hipMemcpyDeviceToHost);

	for(i = 0; i < Nd; i++ )
	{
		Qrs_x[i]      = Qrs_x[i]      * Lambdas.Lambda_q_sqrt[i];
		Qrs_x[i + Nd] = Qrs_x[i + Nd] * Lambdas.Lambda_q_sqrt[i];
	}

	hipMemcpy(Nd2_temp, Qrs_x, dsizeNd2, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Pr(cuHandles, fft_buffer, Nd2_temp, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_k, Lambdas_cuda.dPi_Pr, N3_temp1);
	}
	else
	{
		FAME_Matrix_Vector_Production_Pr(cuHandles, fft_buffer, Nd2_temp, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Pr, N3_temp1);
	}

	hipMemcpy(vec_y, N3_temp1, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);

	mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_x, N12, Ele_field_mtx_N);
	

	cmpx* test = (cmpx*) malloc(Ele_field_mtx_N * sizeof(cmpx));
	vec_plus(test, 1.0, vec_temp, -1.0, vec_y, Ele_field_mtx_N);
	double SVD_test_C = vec_norm(test, Ele_field_mtx_N);

	printf("          EigDecomp_test_C1 = %e\n", C1_error);
    printf("          EigDecomp_test_C2 = %e\n", C2_error);
    printf("          EigDecomp_test_C3 = %e\n", C3_error);
	printf("          SVD_test_C        = %e\n", SVD_test_C);

	if(C1_error > 1e-6 || C2_error > 1e-6 || C3_error > 1e-6 || SVD_test_C > 1e-6)
	{
		printf("\033[40;31mFAME_Main_Code(330):\033[0m\n");
        printf("\033[40;31mThe eigen decomposition is not correct.\033[0m\n");
        printf("\033[40;31mIf N = Nx * Ny * Nz > 256^3, may be caused by numerical errors, please loosen 1e-6.\n");
        printf("\033[40;31mIf not, please contact us.\033[0m\n");
        assert(0);
	}
	
	hipFree(Nd2_temp);
	free(test); free(vec_temp); free(Qrs_x);
	free(vec_x); free(vec_y);
}

void Check_Residual(double* Freq_array, cmpx* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant)
{
	int Ele_field_mtx_N = N * 3;
	int N12 = N * 12;
	size_t size;

	size = Ele_field_mtx_N * Nwant * sizeof(cmpx);

	cmpx* vec_temp = (cmpx*)malloc(size);
	cmpx* vec_left = (cmpx*)malloc(size);
	cmpx* residual = (cmpx*)malloc(size);

	double res, omega2;
	double* B_eps = (double*)calloc(Ele_field_mtx_N, sizeof(double));
	checkCudaErrors(hipMemcpy(B_eps, mtx_B.B_eps, Ele_field_mtx_N*sizeof(double), hipMemcpyDeviceToHost));

	for(int i = 0; i < Nwant; i++)
	{
		omega2 = -pow(Freq_array[i], 2);
		mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*Ele_field_mtx_N, N12, Ele_field_mtx_N);
		mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_temp, N12, Ele_field_mtx_N, "Conjugate Transpose");
		mtx_dot_prod(B_eps, Ele_field_mtx + i*Ele_field_mtx_N, vec_temp, Ele_field_mtx_N, 1);
		vec_plus(residual, 1.0, vec_left, omega2, vec_temp, Ele_field_mtx_N);

		res = vec_norm(residual, Ele_field_mtx_N);

		printf("                 ");
		if(res > 1e-9)
		{
			printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
			// Freq_array[i] = -Freq_array[i];
		}
		else
			printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
	}

	free(vec_temp); free(vec_left); free(residual);
	free(B_eps);
}

void Check_Residual_Biiso(double* Freq_array, cmpx* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant)
{
	int mtx_N = N * 6;
	int N3 = N * 3;
	int N12 = N * 12;
	size_t size;
	
	size = mtx_N * Nwant * sizeof(cmpx);
	cmpx scal=0.0+1.0*I;

	cmpx* vec_temp = (cmpx*)malloc(size);
	cmpx* vec_left = (cmpx*)malloc(size);
	cmpx* residual = (cmpx*)malloc(size);

	double res, omega2;
	double* B_eps = (double*)calloc(N3, sizeof(double));
	cmpx* B_zeta = (cmpx*)calloc(N3, sizeof(cmpx));
	double* B_mu = (double*)calloc(N3, sizeof(double));
	cmpx* B_xi = (cmpx*)calloc(N3, sizeof(cmpx));
	checkCudaErrors(hipMemcpy(B_eps, mtx_B.B_eps, N3*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_zeta, mtx_B.B_zeta, N3*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_mu, mtx_B.B_mu, N3*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_xi, mtx_B.B_xi, N3*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

	
	for(int i = 0; i < Nwant; i++)
	{
		omega2 = -Freq_array[i];

		mtx_dot_prod(B_zeta, Ele_field_mtx + i*mtx_N, residual, N3, 1);
		mtx_dot_prod(B_mu, Ele_field_mtx + i*mtx_N + N3, vec_left, N3, 1);
		mtx_dot_prod(B_eps, Ele_field_mtx + i*mtx_N, residual + N3, N3, 1);
		mtx_dot_prod(B_xi, Ele_field_mtx + i*mtx_N + N3, vec_left + N3, N3, 1);


		vec_plus(vec_temp, scal, vec_left, scal, residual, N3);
		vec_plus(vec_temp + N3, -scal, vec_left + N3, -scal, residual + N3, N3);

		mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N, N12, N3);
		mtx_prod(vec_left + N3, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N + N3, N12, N3, "Conjugate Transpose");
			
		vec_plus(residual, 1.0, vec_left, omega2, vec_temp, mtx_N);

		res = vec_norm(residual, mtx_N);

		printf("                 ");
		if(res > 1e-9)
		{
			printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
			//Freq_array[i] = -Freq_array[i];
		}
		else
			printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
	}

	free(vec_temp); free(vec_left); free(residual);
	free(B_eps);free(B_zeta);free(B_xi);free(B_mu);
	
}