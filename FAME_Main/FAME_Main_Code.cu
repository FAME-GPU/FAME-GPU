#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "FAME_FFT_CUDA.cuh"
#include "FAME_Create_cublas.cuh"
#include "FAME_Create_Buffer.cuh"
#include "FAME_Create_Buffer_Biisotropic.cuh"
#include "FAME_Matrix_B_Isotropic.cuh"
#include "FAME_Malloc_mtx_C.h"
#include "FAME_Matrix_Lambdas.cuh"
#include "FAME_Matrix_Curl.h"
#include "FAME_Create_Frequency_txt.h"
#include "FAME_Save_Eigenvector.h"
#include "FAME_Profile.h"
#include "FAME_Destroy_Main.cuh"
#include "FAME_Fast_Algorithms_Isotropic.cuh"
#include "FAME_Fast_Algorithms_Biisotropic.cuh"
#include "FAME_Matrix_B_Biisotropic.cuh"
#include "FAME_Matrix_Vector_Production_Qrs.cuh"
#include "FAME_Matrix_Vector_Production_Pr.cuh"
#include "FAME_Print_Parameter.h"
#include "FAME_Create_Lambdas_txt.h"
#include "vec_plus.h"
#include "vec_norm.h"
#include "vec_inner_prod.h"
#include "mtx_print.h"
#include "mtx_prod.h"
#include "mtx_trans.h"
#include "mtx_trans_conj.h"
#include "mtx_cat.h"
#include "mtx_dot_prod.h"
#include "kron_vec.h"
#include "inv3.h"

void FAME_Fast_Algorithms_Driver(
	realCPU*        Freq_array,
	cmpxCPU*          Ele_field_mtx,
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	MATERIAL 	   material,
	ES             es,
	LS             ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* lattice_type, PROFILE* Profile);

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type);
void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, realCPU error, PROFILE* Profile);
void Check_Residual(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant);
void Check_Residual_Biiso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant);

int FAME_Main_Code(PAR Par, PROFILE* Profile)
{
	int Nx = Par.mesh.grid_nums[0];
    int Ny = Par.mesh.grid_nums[1];
	int Nz = Par.mesh.grid_nums[2];
	int Nd;
	int N  = Nx * Ny * Nz;
	int Ele_field_mtx_N=0;
	int Nwant = Par.es.nwant;
	int Nstep = Par.es.nstep;
	int N_wave_vec = Par.recip_lattice.Wave_vec_num;
	realCPU wave_vec_array[3];
	
	#if defined(USE_SINGLE)
		Par.ce_error = 1e-4;
		Par.es.tol = 1e-6;
		Par.ls.tol = 1e-6;
	#else
		Par.ce_error = 1e-10;
		Par.es.tol = 1e-12;
		Par.ls.tol = 1e-12;
	#endif 

	realCPU accum;
	struct timespec start, end;

	hipSetDevice(Par.flag.device);
	
    CULIB_HANDLES  cuHandles;
	FFT_BUFFER     fft_buffer;
	LANCZOS_BUFFER lBuffer;
	MTX_B          mtx_B;
	MTX_C          mtx_C;
	LAMBDAS        Lambdas;
    LAMBDAS_CUDA   Lambdas_cuda;

	FAME_Create_cublas(&cuHandles, Nx, Ny, Nz);


	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		Ele_field_mtx_N = N * 3;
	}
	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
	{
		Ele_field_mtx_N = N * 6;
	}
	realCPU* Freq_array    = (realCPU*) calloc(N_wave_vec * Nwant, sizeof(realCPU));
	cmpxCPU*   Ele_field_mtx = (cmpxCPU*)   calloc(Ele_field_mtx_N * Nwant, sizeof(cmpxCPU));
	
	
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		FAME_Create_Buffer(&cuHandles, &fft_buffer, &lBuffer, N, Nstep);
	}
	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
	{
		FAME_Create_Buffer_Biisotropic(&cuHandles, &fft_buffer, &lBuffer,N, Nstep);
	}

	
	if(strcmp(Par.material.material_type, "isotropic") == 0)
	{
		printf("= = = = FAME_Matrix_B_Isotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		size_t size = 3*N*sizeof(realCPU);
		checkCudaErrors(hipMalloc((void**) &mtx_B.B_eps,    size));
		checkCudaErrors(hipMalloc((void**) &mtx_B.invB_eps, size));
		FAME_Matrix_B_Isotropic(mtx_B.B_eps, mtx_B.invB_eps, Par.material, N);
	}

	else if(strcmp(Par.material.material_type, "biisotropic") == 0)
    {
		printf("= = = = FAME_Matrix_B_Biisotropic = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		size_t  size = 3*N*sizeof(realCPU);
		hipMalloc((void**) &mtx_B.B_eps, size);
		hipMalloc((void**) &mtx_B.B_mu, size);
		hipMalloc((void**) &mtx_B.invPhi,  size);

		size = 3*N*sizeof(cmpxGPU);
		hipMalloc((void**) &mtx_B.B_zeta, size);
		hipMalloc((void**) &mtx_B.B_zeta_s, size);
		hipMalloc((void**) &mtx_B.B_xi, size);

		FAME_Matrix_B_Biisotropic(N, Par.material, mtx_B.B_eps, mtx_B.B_mu, mtx_B.B_xi, mtx_B.B_zeta, mtx_B.B_zeta_s, mtx_B.invPhi );

	}


    FAME_Malloc_mtx_C(&mtx_C, N);
	FAME_Print_Parameter(Par );
	for(int i = 0; i < N_wave_vec; i++)
    //for(int i = 0; i < 1; i++)
	{
		Profile->idx = i;

		wave_vec_array[0] = Par.recip_lattice.WaveVector[3 * i];
    	wave_vec_array[1] = Par.recip_lattice.WaveVector[3 * i + 1];
    	wave_vec_array[2] = Par.recip_lattice.WaveVector[3 * i + 2];

    	printf("\033[40;33m= = Start to compute (%3d/%3d) WaveVector = [ % .6f % .6f % .6f ] = =\033[0m\n", i + 1, Par.recip_lattice.Wave_vec_num, wave_vec_array[0], wave_vec_array[1], wave_vec_array[2]);

		printf("= = = = FAME_Matrix_Curl  = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		FAME_Matrix_Curl(&mtx_C, wave_vec_array, Par.mesh.grid_nums, Par.mesh.edge_len, Par.mesh.mesh_len, Par.lattice);

		printf("= = = = FAME_Matrix_Lambdas = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		Nd = FAME_Matrix_Lambdas(&Lambdas_cuda, wave_vec_array, Par.mesh.grid_nums, Par.mesh.mesh_len, Par.lattice.lattice_vec_a, &Par, &Lambdas);
		

		printf("= = = = Check_Eigendecomp = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime(CLOCK_REALTIME, &start);
		Check_Eigendecomp(mtx_C, Lambdas, Lambdas_cuda, fft_buffer, cuHandles, Nx, Ny, Nz, Nd, N, Par.lattice.lattice_type, Par.ce_error, Profile);
		clock_gettime(CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);

		printf("= = = = FAME_Fast_Algorithms = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);
		FAME_Fast_Algorithms_Driver(Freq_array+i*Nwant, Ele_field_mtx, 
			cuHandles, lBuffer, fft_buffer, Lambdas_cuda, mtx_B, Par.material, Par.es, Par.ls,
			Nx, Ny, Nz, Nd, N, Par.lattice.lattice_type, Profile);
		clock_gettime (CLOCK_REALTIME, &end);
		Profile->es_time[Profile->idx] = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;
		
		printf("= = = = Check_Residual  = = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
		clock_gettime (CLOCK_REALTIME, &start);
		if(strcmp(Par.material.material_type, "isotropic") == 0)
		{
			Check_Residual(Freq_array+i*Nwant, Ele_field_mtx, mtx_B, mtx_C, N, Nwant);
		}
		else if(strcmp(Par.material.material_type, "biisotropic") == 0)
		{
			Check_Residual_Biiso(Freq_array+i*Nwant, Ele_field_mtx, mtx_B, mtx_C, N, Nwant);
		}
		//getchar();
		clock_gettime (CLOCK_REALTIME, &end);
		accum = ( end.tv_sec - start.tv_sec ) + ( end.tv_nsec - start.tv_nsec ) / BILLION;
		printf("%*s%8.2f sec.\n", 68, "", accum);

		if(Par.flag.save_eigen_vector)
		{
			printf("= = = = Save Eigen Vector = = = = = = = = = = = = = = = = = = = = = = = = = = = =\n");
			FAME_Save_Eigenvector(Ele_field_mtx, Nwant, Ele_field_mtx_N, i);
		}

		Destroy_Lambdas(Lambdas, Lambdas_cuda, Par.lattice.lattice_type);

		FAME_Print_Profile(*Profile);

	}

	FAME_Create_Frequency_txt(Freq_array, Nwant, Profile->idx);
	
	FAME_Destroy_Main(cuHandles, fft_buffer, lBuffer, mtx_B, mtx_C, Freq_array, Ele_field_mtx);
	
	return 0;
}

void FAME_Fast_Algorithms_Driver(
	realCPU*        Freq_array,
	cmpxCPU*          Ele_field_mtx,
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	MATERIAL 	   material,
	ES             es,
	LS             ls,
	int Nx, int Ny, int Nz, int Nd, int N, 
	char* lattice_type, PROFILE* Profile)
{

	if(strcmp(material.material_type, "isotropic") == 0)
	{
		if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
		{

			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								  Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "Simple", Profile);
		}
		else
		{
			FAME_Fast_Algorithms_Isotropic(Freq_array, Ele_field_mtx, cuHandles, lBuffer, fft_buffer,
								 Lambdas_cuda, mtx_B, es, ls, Nx, Ny, Nz, Nd, N, "General", Profile);
		}
	}
	else if (strcmp(material.material_type, "biisotropic") == 0)
	{
		if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
			(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
			(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
		{		
			FAME_Fast_Algorithms_Biisotropic(Freq_array, Ele_field_mtx, cuHandles, Lambdas_cuda, lBuffer,fft_buffer, mtx_B,material ,Nx, Ny, Nz, Nd,
												es, ls, "Simple", Profile);		
		}
		else
		{
			FAME_Fast_Algorithms_Biisotropic(Freq_array, Ele_field_mtx, cuHandles,Lambdas_cuda, lBuffer, fft_buffer, mtx_B,material ,Nx, Ny, Nz, Nd,
				es, ls , "General",Profile);			
		}
	}
}

void Destroy_Lambdas(LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, char* lattice_type)
{
	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
        free(Lambdas.D_k);
        free(Lambdas.D_ks);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_k);
		hipFree(Lambdas_cuda.dD_ks);
	}

	else
	{
        free(Lambdas.D_kx);
        free(Lambdas.D_ky);
        free(Lambdas.D_kz);
        free(Lambdas.Lambda_x);
        free(Lambdas.Lambda_y);
        free(Lambdas.Lambda_z);

		hipFree(Lambdas_cuda.dD_kx);
    	hipFree(Lambdas_cuda.dD_ky);
    	hipFree(Lambdas_cuda.dD_kz);
	}

    free(Lambdas.Lambda_q_sqrt);
    free(Lambdas.Pi_Qr);
    free(Lambdas.Pi_Pr);
    free(Lambdas.Pi_Qrs);
    free(Lambdas.Pi_Prs);

    hipFree(Lambdas_cuda.Lambda_q_sqrt);
	hipFree(Lambdas_cuda.dPi_Qr);
	hipFree(Lambdas_cuda.dPi_Pr);
	hipFree(Lambdas_cuda.dPi_Qrs);
	hipFree(Lambdas_cuda.dPi_Prs);
}

void Check_Eigendecomp(MTX_C mtx_C, LAMBDAS Lambdas, LAMBDAS_CUDA Lambdas_cuda, FFT_BUFFER fft_buffer, CULIB_HANDLES cuHandles,
	int Nx, int Ny, int Nz, int Nd, int N, char* lattice_type, realCPU error, PROFILE* Profile)
{
	int i;
	int N2 = N * 2;
	int Ele_field_mtx_N = N * 3;
	int N12 = N * 12;
	size_t size, dsizeEle_field_mtx_N, dsizeNd2;

	size = Ele_field_mtx_N * sizeof(cmpxCPU);

	cmpxCPU* vec_x    = (cmpxCPU*) malloc(size);
	cmpxCPU* vec_y    = (cmpxCPU*) malloc(size);
	cmpxCPU* vec_temp = (cmpxCPU*) malloc(size);

	cmpxGPU* N3_temp1 = cuHandles.N3_temp1;
	cmpxGPU* N3_temp2 = cuHandles.N3_temp2;

	cmpxGPU* Nd2_temp;
	dsizeEle_field_mtx_N = Ele_field_mtx_N * sizeof(cmpxGPU);
	dsizeNd2 = Nd * 2 * sizeof(cmpxGPU);

	checkCudaErrors(hipMalloc((void**)&Nd2_temp, dsizeNd2));

	srand(time(NULL));

	for(i = 0; i < Ele_field_mtx_N; i++)
	//vec_x[i] = ((realCPU) rand()/(RAND_MAX + 1.0))  for test
		vec_x[i] = ((realCPU) rand()/(RAND_MAX + 1.0)) +  I * ((realCPU) rand()/(RAND_MAX + 1.0));

	cmpxCPU *vec_y_1, *vec_y_2, *vec_y_3;

	hipMemcpy(N3_temp1, vec_x, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if( (strcmp(lattice_type, "simple_cubic"          ) == 0) || \
		(strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
		(strcmp(lattice_type, "primitive_tetragonal"  ) == 0) )
	{
		//FFT_CUDA(cuHandles, fft_buffer, N3_temp1, Nx, Ny, Nz, Lambdas_cuda.dD_ks, N3_temp2);
		FFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_ks, fft_buffer, cuHandles, Nx, Ny, Nz);
	}
	else
	{
		for(i = 0; i < 3; i++)
        	spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, -1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);
	vec_y_1 = &vec_y[0];  vec_y_2 = &vec_y[N];  vec_y_3 = &vec_y[N2];

	if(Nd == N - 1)
	{
		vec_y_1[0] = 0; vec_y_2[0] = 0; vec_y_3[0] = 0;
		for(i = 0; i < N - 1; i++)
		{
			vec_y_1[i + 1] = Lambdas.Lambda_x[i] * vec_y_1[i + 1];
			vec_y_2[i + 1] = Lambdas.Lambda_y[i] * vec_y_2[i + 1];
			vec_y_3[i + 1] = Lambdas.Lambda_z[i] * vec_y_3[i + 1];
		}
	}
	else
	{
		for(i = 0; i < N; i++)
		{
			vec_y_1[i] = Lambdas.Lambda_x[i] * vec_y_1[i];
			vec_y_2[i] = Lambdas.Lambda_y[i] * vec_y_2[i];
			vec_y_3[i] = Lambdas.Lambda_z[i] * vec_y_3[i];
		}
	}

	hipMemcpy(N3_temp1, vec_y, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		IFFT_CUDA(N3_temp2, N3_temp1, Lambdas_cuda.dD_k, fft_buffer, cuHandles, Nx, Ny, Nz);
		//IFFT_CUDA(cuHandles, fft_buffer, N3_temp1, Nx, Ny, Nz, Lambdas_cuda.dD_k, N3_temp2);
	}
	else
	{
		for(i = 0; i < 3; i++)
			spMV_fastT_gpu(N3_temp2+i*N, N3_temp1+i*N, cuHandles, &fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Nx, Ny, Nz, 1);
	}

	hipMemcpy(vec_y, N3_temp2, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);

	mtx_prod(&vec_temp[0] , mtx_C.C1_r, mtx_C.C1_c, mtx_C.C1_v, &vec_x[0] , N2, N);
	mtx_prod(&vec_temp[N] , mtx_C.C2_r, mtx_C.C2_c, mtx_C.C2_v, &vec_x[N] , N2, N);
	mtx_prod(&vec_temp[N2], mtx_C.C3_r, mtx_C.C3_c, mtx_C.C3_v, &vec_x[N2], N2, N);

	size = N * sizeof(cmpxCPU);
	cmpxCPU* test_x = (cmpxCPU*) malloc(size);
	cmpxCPU* test_y = (cmpxCPU*) malloc(size);
	cmpxCPU* test_z = (cmpxCPU*) malloc(size);

	vec_plus(test_x, 1.0, &vec_temp[0] , -1.0, &vec_y[0] , N);
	vec_plus(test_y, 1.0, &vec_temp[N] , -1.0, &vec_y[N] , N);
	vec_plus(test_z, 1.0, &vec_temp[N2], -1.0, &vec_y[N2], N);
	
	realCPU C1_error = vec_norm(test_x, N);
    realCPU C2_error = vec_norm(test_y, N);
    realCPU C3_error = vec_norm(test_z, N);

	free(test_x); free(test_y); free(test_z);

	cmpxCPU* Qrs_x = (cmpxCPU*) malloc(2*Nd*sizeof(cmpxCPU));

	hipMemcpy(N3_temp1, vec_x, dsizeEle_field_mtx_N, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Lambdas_cuda.dD_ks, Lambdas_cuda.dPi_Qrs, Nx, Ny, Nz, Nd, Profile);
	}
	else
	{
		FAME_Matrix_Vector_Production_Qrs(Nd2_temp, N3_temp1, cuHandles, fft_buffer, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Qrs, Nx, Ny, Nz, Nd, Profile);
	}

	hipMemcpy(Qrs_x, Nd2_temp, dsizeNd2, hipMemcpyDeviceToHost);

	for(i = 0; i < Nd; i++ )
	{
		Qrs_x[i]      = Qrs_x[i]      * Lambdas.Lambda_q_sqrt[i];
		Qrs_x[i + Nd] = Qrs_x[i + Nd] * Lambdas.Lambda_q_sqrt[i];
	}

	hipMemcpy(Nd2_temp, Qrs_x, dsizeNd2, hipMemcpyHostToDevice);

	if((strcmp(lattice_type, "simple_cubic"          ) == 0) || \
	   (strcmp(lattice_type, "primitive_orthorhombic") == 0) || \
	   (strcmp(lattice_type, "primitive_tetragonal"  ) == 0))
	{
		FAME_Matrix_Vector_Production_Pr(cuHandles, fft_buffer, Nd2_temp, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_k, Lambdas_cuda.dPi_Pr, N3_temp1);
	}
	else
	{
		FAME_Matrix_Vector_Production_Pr(cuHandles, fft_buffer, Nd2_temp, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Pr, N3_temp1);
	}

	hipMemcpy(vec_y, N3_temp1, dsizeEle_field_mtx_N, hipMemcpyDeviceToHost);

	mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_x, N12, Ele_field_mtx_N);
	

	cmpxCPU* test = (cmpxCPU*) malloc(Ele_field_mtx_N * sizeof(cmpxCPU));
	vec_plus(test, 1.0, vec_temp, -1.0, vec_y, Ele_field_mtx_N);
	realCPU SVD_test_C = vec_norm(test, Ele_field_mtx_N);

	printf("          EigDecomp_test_C1 = %e\n", C1_error);
    printf("          EigDecomp_test_C2 = %e\n", C2_error);
    printf("          EigDecomp_test_C3 = %e\n", C3_error);
	printf("          SVD_test_C        = %e\n", SVD_test_C);


	if(C1_error > error || C2_error > error || C3_error > error || SVD_test_C > error)
	{
		printf("\033[40;31mFAME_Main_Code(330):\033[0m\n");
		printf("\033[40;31mThe eigen decomposition is not correct.\033[0m\n");
		printf("\033[40;31mIf N = Nx * Ny * Nz > 256^3, may be caused by numerical errors, please loosen 1e-6.\n");
		printf("\033[40;31mIf not, please contact us.\033[0m\n");
		assert(0);
	}
	
	hipFree(Nd2_temp);
	free(test); free(vec_temp); free(Qrs_x);
	free(vec_x); free(vec_y);
}

void Check_Residual(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant)
{
	int Ele_field_mtx_N = N * 3;
	int N12 = N * 12;
	size_t size;

	size = Ele_field_mtx_N * Nwant * sizeof(cmpxCPU);

	cmpxCPU* vec_temp = (cmpxCPU*)malloc(size);
	cmpxCPU* vec_left = (cmpxCPU*)malloc(size);
	cmpxCPU* residual = (cmpxCPU*)malloc(size);

	realCPU res, omega2;
	realCPU* B_eps = (realCPU*)calloc(Ele_field_mtx_N, sizeof(realCPU));
	checkCudaErrors(hipMemcpy(B_eps, mtx_B.B_eps, Ele_field_mtx_N*sizeof(realCPU), hipMemcpyDeviceToHost));
	
	int single;
	#if defined(USE_SINGLE)
	single=1;
	#else
	single=0;
	#endif 

	if (single==1)
	{
		for(int i = 0; i < Nwant; i++)
		{
			omega2 = -pow(Freq_array[i], 2);
			mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*Ele_field_mtx_N, N12, Ele_field_mtx_N);
			mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_temp, N12, Ele_field_mtx_N, "Conjugate Transpose");
			mtx_dot_prod(B_eps, Ele_field_mtx + i*Ele_field_mtx_N, vec_temp, Ele_field_mtx_N, 1);
			vec_plus(residual, 1.0, vec_left, omega2, vec_temp, Ele_field_mtx_N);

			res = vec_norm(residual, Ele_field_mtx_N);

			printf("                 ");
			if(res > 1e-4)
			{
				printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
				// Freq_array[i] = -Freq_array[i];
			}
			else
				printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
		}	
	}
	else
	{
		for(int i = 0; i < Nwant; i++)
		{
			omega2 = -pow(Freq_array[i], 2);
			mtx_prod(vec_temp, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*Ele_field_mtx_N, N12, Ele_field_mtx_N);
			mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, vec_temp, N12, Ele_field_mtx_N, "Conjugate Transpose");
			mtx_dot_prod(B_eps, Ele_field_mtx + i*Ele_field_mtx_N, vec_temp, Ele_field_mtx_N, 1);
			vec_plus(residual, 1.0, vec_left, omega2, vec_temp, Ele_field_mtx_N);

			res = vec_norm(residual, Ele_field_mtx_N);

			printf("                 ");
			if(res > 1e-9)
			{
				printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
				// Freq_array[i] = -Freq_array[i];
			}
			else
				printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
		}	
	}

	free(vec_temp); free(vec_left); free(residual);
	free(B_eps);
}

void Check_Residual_Biiso(realCPU* Freq_array, cmpxCPU* Ele_field_mtx, MTX_B mtx_B, MTX_C mtx_C, int N, int Nwant)
{
	int mtx_N = N * 6;
	int N3 = N * 3;
	int N12 = N * 12;
	size_t size;
	
	size = mtx_N * Nwant * sizeof(cmpxCPU);
	cmpxCPU scal=0.0+1.0*I;

	cmpxCPU* vec_temp = (cmpxCPU*)malloc(size);
	cmpxCPU* vec_left = (cmpxCPU*)malloc(size);
	cmpxCPU* residual = (cmpxCPU*)malloc(size);

	realCPU res, omega2;
	realCPU* B_eps = (realCPU*)calloc(N3, sizeof(realCPU));
	cmpxCPU* B_zeta = (cmpxCPU*)calloc(N3, sizeof(cmpxCPU));
	realCPU* B_mu = (realCPU*)calloc(N3, sizeof(realCPU));
	cmpxCPU* B_xi = (cmpxCPU*)calloc(N3, sizeof(cmpxCPU));
	checkCudaErrors(hipMemcpy(B_eps, mtx_B.B_eps, N3*sizeof(realCPU), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_zeta, mtx_B.B_zeta, N3*sizeof(cmpxGPU), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_mu, mtx_B.B_mu, N3*sizeof(realCPU), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(B_xi, mtx_B.B_xi, N3*sizeof(cmpxGPU), hipMemcpyDeviceToHost));

	int single;
	#if defined(USE_SINGLE)
	single=1;
	#else
	single=0;
	#endif 

	if (single==1)
	{
		for(int i = 0; i < Nwant; i++)
		{
			omega2 = -Freq_array[i];

			mtx_dot_prod(B_zeta, Ele_field_mtx + i*mtx_N, residual, N3, 1);
			mtx_dot_prod(B_mu, Ele_field_mtx + i*mtx_N + N3, vec_left, N3, 1);
			mtx_dot_prod(B_eps, Ele_field_mtx + i*mtx_N, residual + N3, N3, 1);
			mtx_dot_prod(B_xi, Ele_field_mtx + i*mtx_N + N3, vec_left + N3, N3, 1);


			vec_plus(vec_temp, scal, vec_left, scal, residual, N3);
			vec_plus(vec_temp + N3, -scal, vec_left + N3, -scal, residual + N3, N3);

			mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N, N12, N3);
			mtx_prod(vec_left + N3, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N + N3, N12, N3, "Conjugate Transpose");
			
			vec_plus(residual, 1.0, vec_left, omega2, vec_temp, mtx_N);

			res = vec_norm(residual, mtx_N);

			printf("                 ");
			if(res > 1e-4)
			{
				printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
				//Freq_array[i] = -Freq_array[i];
			}
			else
				printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
		}
	}
	else
	{
		for(int i = 0; i < Nwant; i++)
		{
			omega2 = -Freq_array[i];

			mtx_dot_prod(B_zeta, Ele_field_mtx + i*mtx_N, residual, N3, 1);
			mtx_dot_prod(B_mu, Ele_field_mtx + i*mtx_N + N3, vec_left, N3, 1);
			mtx_dot_prod(B_eps, Ele_field_mtx + i*mtx_N, residual + N3, N3, 1);
			mtx_dot_prod(B_xi, Ele_field_mtx + i*mtx_N + N3, vec_left + N3, N3, 1);


			vec_plus(vec_temp, scal, vec_left, scal, residual, N3);
			vec_plus(vec_temp + N3, -scal, vec_left + N3, -scal, residual + N3, N3);

			mtx_prod(vec_left, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N, N12, N3);
			mtx_prod(vec_left + N3, mtx_C.C_r, mtx_C.C_c, mtx_C.C_v, Ele_field_mtx + i*mtx_N + N3, N12, N3, "Conjugate Transpose");
			
			vec_plus(residual, 1.0, vec_left, omega2, vec_temp, mtx_N);

			res = vec_norm(residual, mtx_N);

			printf("                 ");
			if(res > 1e-9)
			{
				printf("\033[40;31mFreq(%2d) = %10.8f, residual = %e.\033[0m\n", i, Freq_array[i], res);
				//Freq_array[i] = -Freq_array[i];
			}
			else
				printf("Freq(%2d) = %10.8f, residual = %e.\n", i, Freq_array[i], res);
		}
	}

	free(vec_temp); free(vec_left); free(residual);
	free(B_eps);free(B_zeta);free(B_xi);free(B_mu);
	
}