#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "Lanczos_Isotropic.cuh"
#include "FAME_Matrix_Vector_Production_Qr.cuh"
#include "printDeviceArray.cuh"

static __global__ void initialize_iso(cmpxGPU* vec, realCPU real, realCPU imag, int size);
static __global__ void dot_product(cmpxGPU* vec_y, realCPU* array, int size);


int Eigen_Restoration_Isotropic(
	cmpxGPU* Output_eigvec_mat,
	cmpxGPU* Input_eigvec_mat,
	CULIB_HANDLES    cuHandles,
	FFT_BUFFER       fft_buffer,
	LAMBDAS_CUDA     Lambdas_cuda,
	MTX_B            mtx_B,
	int Nx, int Ny, int Nz, int Nd, int N, int Nwant,
	string flag_CompType, PROFILE* Profile);

int FAME_Fast_Algorithms_Isotropic(
	realCPU*        Freq_array,
	cmpxCPU*          Ele_field_mtx,
	CULIB_HANDLES  cuHandles,
	LANCZOS_BUFFER lBuffer,
	FFT_BUFFER     fft_buffer,
	LAMBDAS_CUDA   Lambdas_cuda,
	MTX_B          mtx_B,
	ES 			   es,
	LS 			   ls,
	int Nx, int Ny, int Nz, int Nd, int N,
	string flag_CompType, PROFILE* Profile)
{
	int N3 = 3 * N;
	int Nd2 = Nd * 2;
	size_t memsize;
	// Creat temp vector
	memsize = Nd2 * (es.nstep + 1) * sizeof(cmpxGPU);
	checkCudaErrors(hipMalloc((void**) &lBuffer.dU, memsize));

	cmpxGPU* ev;
	checkCudaErrors(hipMalloc((void**)&ev, Nd * 2 * ( es.nwant+2 ) * sizeof(cmpxGPU)));

    memsize = Nd2 * sizeof(cmpxGPU);
    checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp1, memsize));
    checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp2, memsize));
    checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp3, memsize));
	checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp4, memsize));

    Lanczos_Isotropic(Freq_array, ev, cuHandles, lBuffer, fft_buffer, Lambdas_cuda, mtx_B, es, ls,
					   	                                  Nx, Ny, Nz, Nd, flag_CompType, Profile);

    hipFree(lBuffer.dU);

    cmpxGPU* ev_back;
    checkCudaErrors(hipMalloc((void**)&ev_back, N3 * es.nwant * sizeof(cmpxGPU)));


	Eigen_Restoration_Isotropic(ev_back, ev, cuHandles, fft_buffer, Lambdas_cuda, mtx_B, 
						           Nx, Ny, Nz, Nd, N, es.nwant, flag_CompType, Profile);

	if(Nd == N-1)
	{
		for(int i = es.nwant - 1; i >= 2 ; i--)
		{
			FAME_cublas_swap(cuHandles.cublas_handle, N3, ev_back + i * N3, 1, ev_back + (i - 2) * N3, 1);
			Freq_array[i] = Freq_array[i - 2];
		}

		Freq_array[0] = 0.0;
		Freq_array[1] = 0.0;

		realCPU temp = 1.0 / sqrt(N3);

		dim3 DimBlock(BLOCK_SIZE, 1, 1);
		dim3 DimGrid((N3-1)/BLOCK_SIZE + 1, 1, 1);

		initialize_iso<<<DimGrid, DimBlock>>>(ev_back,      temp, 0.0, N3);
		initialize_iso<<<DimGrid, DimBlock>>>(ev_back + N3, temp, 0.0, N3);
	}

	checkCudaErrors(hipMemcpy(Ele_field_mtx, ev_back, N3 * es.nwant * sizeof(cmpxGPU), hipMemcpyDeviceToHost));
//	printDeviceArray(ev_back,N3 ,"ev_back.txt");
 //getchar();
	hipFree(ev); hipFree(ev_back);hipFree(lBuffer.dU);
	hipFree(cuHandles.Nd2_temp1); hipFree(cuHandles.Nd2_temp2); hipFree(cuHandles.Nd2_temp3); hipFree(cuHandles.Nd2_temp4);
	return 0;
}



int Eigen_Restoration_Isotropic(
	cmpxGPU* Output_eigvec_mat,
	cmpxGPU* Input_eigvec_mat,
	CULIB_HANDLES    cuHandles,
	FFT_BUFFER       fft_buffer,
	LAMBDAS_CUDA     Lambdas_cuda,
	MTX_B            mtx_B,
	int Nx, int Ny, int Nz, int Nd, int N, int Nwant,
	string flag_CompType, PROFILE* Profile)
{
	int N3 = N * 3;
	int Nd2 = Nd * 2;
	realCPU norm;

	dim3 DimBlock(BLOCK_SIZE, 1, 1);
	dim3 DimGrid((N3-1)/BLOCK_SIZE + 1, 1, 1);
	
	for(int i = 0; i < Nwant; i++)
	{
		
		dot_product<<<DimGrid, DimBlock>>>(Input_eigvec_mat+i*Nd2, Lambdas_cuda.Lambda_q_sqrt, Nd2);

		if (flag_CompType == "Simple")
			FAME_Matrix_Vector_Production_Qr(Output_eigvec_mat+i*N3, Input_eigvec_mat+i*Nd2, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_k, Lambdas_cuda.dPi_Qr );
		
		else if (flag_CompType == "General")
			FAME_Matrix_Vector_Production_Qr(Output_eigvec_mat+i*N3, Input_eigvec_mat+i*Nd2, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, Lambdas_cuda.dPi_Qr );
   

		dot_product<<<DimGrid, DimBlock>>>(Output_eigvec_mat+i*N3, mtx_B.invB_eps, N3);

		FAME_cublas_nrm2(cuHandles.cublas_handle, N3, Output_eigvec_mat+N3*i, 1, &norm);
   norm=1.0/norm;
//   cout<<norm<<endl;
//printDeviceArray(Output_eigvec_mat+i*N3,N3 ,"Output1.txt");
		FAME_cublas_dscal(cuHandles.cublas_handle, N3, &norm, Output_eigvec_mat+N3*i, 1);
//   printDeviceArray(Output_eigvec_mat+i*N3,N3 ,"Output.txt");
//   getchar();
	}
	
	return 0;
}

static __global__ void dot_product(cmpxGPU* vec_y, realCPU* array, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        vec_y[idx].x = vec_y[idx].x * array[idx];
		vec_y[idx].y = vec_y[idx].y * array[idx];
    }

}

static __global__ void initialize_iso(cmpxGPU* vec, realCPU real, realCPU imag, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
	{
		vec[idx].x = real;
		vec[idx].y = imag;
	}
}
