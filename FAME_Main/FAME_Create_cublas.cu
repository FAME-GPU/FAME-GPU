#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"

int FAME_Create_cublas(CULIB_HANDLES* cuHandles, int Nx, int Ny, int Nz)
{
    
    hipblasStatus_t cublasErr;
    hipfftResult    cufftErr;

    cublasErr = hipblasCreate(&cuHandles->cublas_handle);
    assert(cublasErr == HIPBLAS_STATUS_SUCCESS);

    cublasErr = hipblasSetPointerMode(cuHandles->cublas_handle, HIPBLAS_POINTER_MODE_HOST);
    assert(cublasErr == HIPBLAS_STATUS_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_x, Nx, PC_cufft_type, Ny*Nz);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_y, Ny, PC_cufft_type, Nx*Nz);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_z, Nz, PC_cufft_type, Nx*Ny);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan3d(&cuHandles->cufft_plan, Nz, Ny, Nx, PC_cufft_type);
    assert(cufftErr == HIPFFT_SUCCESS);

    return 0;
}

