#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"

int FAME_Create_cublas(CULIB_HANDLES* cuHandles, int Nx, int Ny, int Nz)
{
    
    hipblasStatus_t   cublasErr;
    hipsparseStatus_t cusparseErr; 
    hipfftResult      cufftErr;

    cublasErr = hipblasCreate(&cuHandles->cublas_handle);
    assert(cublasErr == HIPBLAS_STATUS_SUCCESS);

    hipsparseCreate(&cuHandles->cusparse_handle);
    assert(cusparseErr == HIPSPARSE_STATUS_SUCCESS);

    cublasErr = hipblasSetPointerMode(cuHandles->cublas_handle, HIPBLAS_POINTER_MODE_HOST);
    assert(cublasErr == HIPBLAS_STATUS_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_x, Nx, FAME_cufft_type, Ny*Nz);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_y, Ny, FAME_cufft_type, Nx*Nz);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan1d(&cuHandles->cufft_plan_1d_z, Nz, FAME_cufft_type, Nx*Ny);
    assert(cufftErr == HIPFFT_SUCCESS);

    cufftErr = hipfftPlan3d(&cuHandles->cufft_plan, Nz, Ny, Nx, FAME_cufft_type);
    assert(cufftErr == HIPFFT_SUCCESS);

    return 0;
}

