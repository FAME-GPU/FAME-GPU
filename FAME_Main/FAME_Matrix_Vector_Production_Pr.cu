#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include "FAME_FFT_CUDA.cuh"
#include "printDeviceArray.cuh"

static __global__ void vp_add_vp(int size, hipDoubleComplex* L_1, hipDoubleComplex* L_2, hipDoubleComplex* vec_1, hipDoubleComplex* vec_2,hipDoubleComplex* vec_out);

////////////=========================== Create Pr function for Biiso (cuda)===========================//////////////////
int FAME_Matrix_Vector_Production_Pr(   CULIB_HANDLES cuHandles, 
                                        FFT_BUFFER fft_buffer, 
                                        hipDoubleComplex* vec_x, 
                                        int Nx, int Ny, int Nz, int Nd, 
                                        hipDoubleComplex* D_k,
                                        hipDoubleComplex* Pi_Pr, 
                                        hipDoubleComplex* vec_y)
{
    int N = Nx*Ny*Nz;
    //hipDoubleComplex* temp = cuHandles.N3_temp1;
	hipDoubleComplex* temp;
    checkCudaErrors(hipMalloc((void**)&temp, 3*N*sizeof(hipDoubleComplex)));
    dim3 DimBlock(BLOCK_SIZE,1,1);
    dim3 DimGrid((Nd-1)/BLOCK_SIZE +1,1,1);

	// Initial

    checkCudaErrors(hipMemset(temp, 0, N * 3 * sizeof(hipDoubleComplex)));

    vp_add_vp<<<DimGrid, DimBlock>>>(Nd, Pi_Pr,         Pi_Pr+3*Nd, vec_x, vec_x+Nd, temp+N-Nd);
    hipDeviceSynchronize();
    vp_add_vp<<<DimGrid, DimBlock>>>(Nd, Pi_Pr+Nd,      Pi_Pr+4*Nd, vec_x, vec_x+Nd, temp+N-Nd+N);
    hipDeviceSynchronize();
    vp_add_vp<<<DimGrid, DimBlock>>>(Nd, Pi_Pr+2*Nd,    Pi_Pr+5*Nd, vec_x, vec_x+Nd, temp+N-Nd+2*N);
    hipDeviceSynchronize();

  
    IFFT_CUDA(vec_y, temp, D_k, fft_buffer, cuHandles, Nx, Ny, Nz);

	hipFree(temp);

    return 0;
}

int FAME_Matrix_Vector_Production_Pr(CULIB_HANDLES cuHandles, FFT_BUFFER fft_buffer, hipDoubleComplex* vec_x, int Nx, int Ny, int Nz, int Nd, hipDoubleComplex* D_kx, hipDoubleComplex* D_ky, hipDoubleComplex* D_kz, hipDoubleComplex* Pi_Pr, hipDoubleComplex* vec_y)
{
    int N = Nx*Ny*Nz;
    int N3 = N * 3;
    dim3 DimBlock(BLOCK_SIZE,1,1);
    dim3 DimGrid((Nd-1)/BLOCK_SIZE +1,1,1);
    hipDoubleComplex* temp;
    checkCudaErrors(hipMalloc((void**)&temp, N3*sizeof(hipDoubleComplex)));

    checkCudaErrors(hipMemset(temp, 0, N3 * sizeof(hipDoubleComplex)));

    //printDeviceArray( vec_x, 2*Nd, "print_vec_x.txt");
    vp_add_vp<<<DimGrid, DimBlock>>>(Nd, Pi_Pr,         Pi_Pr+3*Nd, vec_x, vec_x+Nd, temp+N-Nd);
    vp_add_vp<<<DimGrid, DimBlock>>>(Nd, Pi_Pr+Nd,      Pi_Pr+4*Nd, vec_x, vec_x+Nd, temp+N-Nd+N);
    vp_add_vp<<<DimGrid, DimBlock>>>(Nd, Pi_Pr+2*Nd,    Pi_Pr+5*Nd, vec_x, vec_x+Nd, temp+N-Nd+2*N);
    
    //printDeviceArray( temp, 3*N, "print_temp.txt");
	for(int i=0; i<3; i++)
        spMV_fastT_gpu( vec_y+i*N, temp+i*N, cuHandles, &fft_buffer, D_kx, D_ky, D_kz, Nx, Ny, Nz, 1);

    hipFree(temp);
    return 0;
}

static __global__ void vp_add_vp(int size, hipDoubleComplex* L_1, hipDoubleComplex* L_2, hipDoubleComplex* vec_1, hipDoubleComplex* vec_2,hipDoubleComplex* vec_out)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < size)
    {
        //vec_out[idx] = L_1[idx]*vec_1[idx] + L_2[idx]*vec_2[idx];
        vec_out[idx].x = L_1[idx].x*vec_1[idx].x + L_2[idx].x*vec_2[idx].x - L_1[idx].y*vec_1[idx].y - L_2[idx].y*vec_2[idx].y;
        vec_out[idx].y = L_1[idx].x*vec_1[idx].y + L_2[idx].y*vec_2[idx].x + L_1[idx].y*vec_1[idx].x + L_2[idx].x*vec_2[idx].y;

    }

}