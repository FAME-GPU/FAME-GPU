#include "hip/hip_runtime.h"
#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include <complex.h>

#include "FAME_Matrix_Vector_Production_Pr.cuh"
#include "FAME_Matrix_Vector_Production_Qr.cuh"
#include "FAME_Matrix_Vector_Production_invB_Biisotropic.cuh"
#include "Lanczos_Biisotropic.cuh"
#include "printDeviceArray.cuh"

//#define BLOCK_SIZE 1024

int Eigen_Restoration_Biisotropic(  CULIB_HANDLES cuHandles,
                                    LAMBDAS_CUDA Lambdas_cuda,
                                    FFT_BUFFER    fft_buffer,
                                    hipDoubleComplex* Input_eigvec_mat,
                                    hipDoubleComplex* Output_eigvec_mat,
                                    MTX_B  mtx_B,
                                    int N_eig_wanted,
                                    int Nx,
                                    int Ny,
                                    int Nz,
                                    int Nd,
                                    std::string flag_CompType, PROFILE* Profile);
 
static __global__ void scaling(int size, hipDoubleComplex* array, double norm_vec);
static __global__ void ones(int size, hipDoubleComplex* array);
static __global__ void initialize(hipDoubleComplex* vec, double real, double imag, int size);

int FAME_Fast_Algorithms_Biisotropic
	(double*        Freq_array,
	 cmpx*          Ele_field_mtx,
	 CULIB_HANDLES cuHandles,
	 LAMBDAS_CUDA  Lambdas_cuda, 
	 LANCZOS_BUFFER lBuffer,
	 FFT_BUFFER    fft_buffer,
	 MTX_B 	mtx_B,
	 MATERIAL material,	 
	 int Nx, 
	 int Ny, 
	 int Nz,
	 int Nd,
	 ES  es,
	 LS  ls, 
	 string flag_CompType,
	 PROFILE* Profile)
{
	cout << "IN FAME_Fast_Algorithms_Biisotropic " << endl;

	int N = Nx*Ny*Nz;
	int Nd4 = Nd * 4;
	int N6 = 6*N;
	dim3 DimBlock(BLOCK_SIZE, 1, 1);
	dim3 DimGrid((N-1)/BLOCK_SIZE+1, 1, 1);
	size_t memsize;
	int eigen_wanted = es.nwant;
	hipblasStatus_t hipblasStatus_t;

	memsize = Nd4 * (es.nstep + 1) * sizeof(hipDoubleComplex);
	checkCudaErrors(hipMalloc((void**) &lBuffer.dU, memsize));
	
	hipDoubleComplex* DEV_Back;
	checkCudaErrors(hipMalloc((void**)&DEV_Back,   sizeof(hipDoubleComplex)*6*N*eigen_wanted));
	hipDoubleComplex* DEV;
	checkCudaErrors(hipMalloc((void**)&DEV,        sizeof(hipDoubleComplex)*4*Nd*(eigen_wanted+2)*2));
	hipDoubleComplex* EW = (hipDoubleComplex*) malloc( eigen_wanted*sizeof(hipDoubleComplex));


	memsize = Nd4 * sizeof(hipDoubleComplex);
    checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp1, memsize));
    checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp2, memsize));
    checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp3, memsize));
	checkCudaErrors(hipMalloc((void**)&cuHandles.Nd2_temp4, memsize));

	
	if (material.chirality_in[0] > sqrt(13))
	{
		cout<<"Chirality_in > sqrt(13), indefinite matrix ！"<<endl;
		assert(0);
	}
	else if (material.chirality_in[0] < sqrt(13))
	{
	
		Lanczos_Biisotropic( cuHandles,
			fft_buffer, lBuffer,
			mtx_B, Nx, Ny, Nz, Nd, es, ls,
			Lambdas_cuda.Lambda_q_sqrt,
			Lambdas_cuda.dPi_Qr,
			Lambdas_cuda.dPi_Pr,
			Lambdas_cuda.dPi_Qrs,
			Lambdas_cuda.dPi_Prs,
			Lambdas_cuda.dD_k,
			Lambdas_cuda.dD_ks,
			Lambdas_cuda.dD_kx,
			Lambdas_cuda.dD_ky,
			Lambdas_cuda.dD_kz,
			Freq_array, DEV,
			flag_CompType,
			Profile );
	}

	hipFree(lBuffer.dU);

	Eigen_Restoration_Biisotropic(  cuHandles, Lambdas_cuda,
		fft_buffer,
		DEV, DEV_Back,
		mtx_B, eigen_wanted,
		Nx, Ny, Nz, Nd,
		flag_CompType, Profile);

	if(Nd == N-1)
	{
			for(int i = es.nwant - 1; i >= 2 ; i--)
			{
				hipblasStatus_t=hipblasZswap(cuHandles.cublas_handle, 6*N, DEV_Back + i * 6*N, 1, DEV_Back + (i - 2) * 6*N, 1);
				assert( hipblasStatus_t == HIPBLAS_STATUS_SUCCESS );
				Freq_array[i] = Freq_array[i - 2];
			}

			Freq_array[0] = 0.0;
			Freq_array[1] = 0.0;

			double temp = 1.0 / sqrt(N6);
			initialize<<<DimGrid, DimBlock>>>(DEV_Back,       temp, 0.0, N);
			initialize<<<DimGrid, DimBlock>>>(DEV_Back + N6,  temp, 0.0, N);


	}

	checkCudaErrors(hipMemcpy(Ele_field_mtx, DEV_Back, N6 * eigen_wanted * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

	hipFree(DEV); hipFree(DEV_Back);
	hipFree(cuHandles.Nd2_temp1); hipFree(cuHandles.Nd2_temp2); hipFree(cuHandles.Nd2_temp3); hipFree(cuHandles.Nd2_temp4);
	return 0;
}

int Eigen_Restoration_Biisotropic( 	CULIB_HANDLES cuHandles,
									LAMBDAS_CUDA Lambdas_cuda,
									FFT_BUFFER    fft_buffer,
									hipDoubleComplex* Input_eigvec_mat, 
									hipDoubleComplex* Output_eigvec_mat,
									MTX_B  mtx_B,	
									int N_eig_wanted, 
									int Nx,
									int Ny,
									int Nz,
									int Nd,
									string flag_CompType,PROFILE* Profile)
{
	int N = Nx*Ny*Nz;
	double norm_vec = 0.0;
	dim3 DimGrid(BLOCK_SIZE, 1, 1);
	dim3 DimBlock((N-1)/BLOCK_SIZE+1, 1, 1);
	hipblasStatus_t hipblasStatus_t;

	hipDoubleComplex* vec_y;
	checkCudaErrors(hipMalloc((void**)&vec_y, 6*N*sizeof(hipDoubleComplex)));

	// Start to restore the eigenvectors

	for(int ii = 0; ii<N_eig_wanted; ii++)
	{
		if( flag_CompType == "Simple" ){
			FAME_Matrix_Vector_Production_Pr( 	cuHandles, 
												fft_buffer, 
												Input_eigvec_mat+4*Nd*ii, 
												Nx, 
												Ny,
												Nz,
												Nd,
												Lambdas_cuda.dD_k,
												Lambdas_cuda.dPi_Pr,
												vec_y);
			
			FAME_Matrix_Vector_Production_Qr( 	vec_y+3*N,
												Input_eigvec_mat+2*Nd+4*Nd*ii,				
												cuHandles, 
												fft_buffer,
												Lambdas_cuda.dD_k,
												Lambdas_cuda.dPi_Qr,
												Nx, Ny, Nz, Nd, Profile );

		}else if( flag_CompType == "General" ){
			//printDeviceArray( Input_eigvec_mat, 2*Nd, "print_Input_eigvec_mat.txt");
			FAME_Matrix_Vector_Production_Pr(	cuHandles, 
												fft_buffer,
												Input_eigvec_mat+4*Nd*ii,
												Nx,
												Ny,
												Nz,
												Nd,
												Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz,
												Lambdas_cuda.dPi_Pr,
												vec_y);									
			//printDeviceArray( vec_y, 3*N, "print_vec_y.txt");												
			FAME_Matrix_Vector_Production_Qr(   vec_y+3*N,
												Input_eigvec_mat+2*Nd+4*Nd*ii,
												cuHandles,
                                                fft_buffer,
                                                Lambdas_cuda.dD_kx, Lambdas_cuda.dD_ky, Lambdas_cuda.dD_kz, 
                                                Lambdas_cuda.dPi_Qr,
                                                Nx, Ny, Nz, Nd, Profile);
		}
	
	FAME_Matrix_Vector_Production_invB_Biisotropic( cuHandles,
                                                    mtx_B,
                                                    N,
                                                    vec_y,
                                                    Output_eigvec_mat+6*N*ii);
	
	//Normalize the eigenvector
	hipblasStatus_t=hipblasDznrm2(cuHandles.cublas_handle, 6*N, Output_eigvec_mat+6*N*ii, 1, &norm_vec );
	assert( hipblasStatus_t == HIPBLAS_STATUS_SUCCESS );
	scaling<<<DimGrid, DimBlock>>>(N, Output_eigvec_mat+6*N*ii, norm_vec);

	}

	hipFree(vec_y);
	return 0;
}



static __global__ void scaling(int size, hipDoubleComplex* array, double norm_vec)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size)
	{
		for( int i=0; i<6; i++)
		{
			array[idx*6+i].x = array[idx*6+i].x/norm_vec;	
			array[idx*6+i].y = array[idx*6+i].y/norm_vec;
		}
	}

}

static __global__ void ones(int size, hipDoubleComplex* array)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;	
	if( idx < size )
	{	for( int i=0; i<6; i++)
		{
			array[idx*6+i].x = 1.0; array[idx*6+i].y = 0.0;
		}
	}

}

static __global__ void initialize(hipDoubleComplex* vec, double real, double imag, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// printf("%d\t", idx);
	if( idx < size )
	{	for( int i=0; i<6; i++)
		{
			vec[idx*6+i].x = real; vec[idx*6+i].y = 0.0;
		}
	}
}











