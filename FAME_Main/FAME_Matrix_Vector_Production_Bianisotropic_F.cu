#include "FAME_Internal_Common.h"
#include "FAME_CUDA.h"
#include <complex.h>

#include "FAME_Matrix_Vector_Production_Qr.cuh"
#include "FAME_Matrix_Vector_Production_Qrs.cuh"
#include "FAME_Matrix_Vector_Production_Bianisotropic_G_Shuffle.cuh"

#include "printDeviceArray.cuh"


void FAME_Matrix_Vector_Production_Bianisotropic_F(CULIB_HANDLES cuHandles, FFT_BUFFER fft_buffer, MTX_B mtx_B, 
         int Nx, int Ny, int Nz, int Nd, 
         cmpxGPU* Pi_Qr, cmpxGPU* Pi_Qrs, 
         cmpxGPU* Pi_Qr_110, cmpxGPU* Pi_Qrs_110,
         cmpxGPU* Pi_Qr_101, cmpxGPU* Pi_Qrs_101, 
         cmpxGPU* Pi_Qr_011, cmpxGPU* Pi_Qrs_011, 
         cmpxGPU* D_k, cmpxGPU* D_ks, 
         cmpxGPU* x, cmpxGPU* y)
{
    int size = 2 * Nd;
    int size_temp = 3 * Nx * Ny * Nz;
    size_t memsize = 8*size_temp*sizeof(cmpxGPU);

    cmpxGPU* temp_vec;
    hipMalloc((void**)&temp_vec, memsize);

    FAME_Matrix_Vector_Production_Qr(temp_vec+0*size_temp, x+0*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr);	
    FAME_Matrix_Vector_Production_Qr(temp_vec+1*size_temp, x+1*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr_110);
    FAME_Matrix_Vector_Production_Qr(temp_vec+2*size_temp, x+2*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr_101);
    FAME_Matrix_Vector_Production_Qr(temp_vec+3*size_temp, x+3*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr_011);

    FAME_Matrix_Vector_Production_Qr(temp_vec+4*size_temp, x+4*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr);
    FAME_Matrix_Vector_Production_Qr(temp_vec+5*size_temp, x+5*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr_110);
    FAME_Matrix_Vector_Production_Qr(temp_vec+6*size_temp, x+6*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr_101);
    FAME_Matrix_Vector_Production_Qr(temp_vec+7*size_temp, x+7*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_k, Pi_Qr_011);

    FAME_Matrix_Vector_Production_Bianisotropic_G_Shuffle(cuHandles, Nx, Ny, Nz, Nd, mtx_B.GInOut_index, mtx_B.GInOut_index_length, mtx_B.G, temp_vec);

    FAME_Matrix_Vector_Production_Qrs(y+0*size, temp_vec+0*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs);
    FAME_Matrix_Vector_Production_Qrs(y+1*size, temp_vec+1*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs_110);
    FAME_Matrix_Vector_Production_Qrs(y+2*size, temp_vec+2*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs_101);
    FAME_Matrix_Vector_Production_Qrs(y+3*size, temp_vec+3*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs_011);

    FAME_Matrix_Vector_Production_Qrs(y+4*size, temp_vec+4*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs);
    FAME_Matrix_Vector_Production_Qrs(y+5*size, temp_vec+5*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs_110);
    FAME_Matrix_Vector_Production_Qrs(y+6*size, temp_vec+6*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs_101);
    FAME_Matrix_Vector_Production_Qrs(y+7*size, temp_vec+7*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_ks, Pi_Qrs_011);

    hipFree( temp_vec );
}


void FAME_Matrix_Vector_Production_Bianisotropic_F(CULIB_HANDLES cuHandles, FFT_BUFFER fft_buffer, MTX_B mtx_B,
    int Nx, int Ny, int Nz, int Nd, 
    cmpxGPU* Pi_Qr, cmpxGPU* Pi_Qrs, 
    cmpxGPU* Pi_Qr_110, cmpxGPU* Pi_Qrs_110,
    cmpxGPU* Pi_Qr_101, cmpxGPU* Pi_Qrs_101, 
    cmpxGPU* Pi_Qr_011, cmpxGPU* Pi_Qrs_011, 
    cmpxGPU* D_kx, cmpxGPU* D_ky, cmpxGPU* D_kz,  
    cmpxGPU* x, cmpxGPU* y)
{
    int size = 2 * Nd;
    int size_temp = 3 * Nx * Ny * Nz;
    size_t memsize = 8 * size_temp * sizeof(cmpxGPU);

    cmpxGPU* temp_vec;
    hipMalloc((void**)&temp_vec, memsize);

    FAME_Matrix_Vector_Production_Qr(temp_vec+0*size_temp, x+0*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr);	
    FAME_Matrix_Vector_Production_Qr(temp_vec+1*size_temp, x+1*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr_110);
    FAME_Matrix_Vector_Production_Qr(temp_vec+2*size_temp, x+2*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr_101);
    FAME_Matrix_Vector_Production_Qr(temp_vec+3*size_temp, x+3*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr_011);

    FAME_Matrix_Vector_Production_Qr(temp_vec+4*size_temp, x+4*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr);
    FAME_Matrix_Vector_Production_Qr(temp_vec+5*size_temp, x+5*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr_110);
    FAME_Matrix_Vector_Production_Qr(temp_vec+6*size_temp, x+6*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr_101);
    FAME_Matrix_Vector_Production_Qr(temp_vec+7*size_temp, x+7*size, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qr_011);

    FAME_Matrix_Vector_Production_Bianisotropic_G_Shuffle(cuHandles, Nx, Ny, Nz, Nd, mtx_B.GInOut_index, mtx_B.GInOut_index_length, mtx_B.G, temp_vec);
    
    FAME_Matrix_Vector_Production_Qrs(y+0*size, temp_vec+0*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs);
    FAME_Matrix_Vector_Production_Qrs(y+1*size, temp_vec+1*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs_110);
    FAME_Matrix_Vector_Production_Qrs(y+2*size, temp_vec+2*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs_101);
    FAME_Matrix_Vector_Production_Qrs(y+3*size, temp_vec+3*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs_011);

    FAME_Matrix_Vector_Production_Qrs(y+4*size, temp_vec+4*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs);
    FAME_Matrix_Vector_Production_Qrs(y+5*size, temp_vec+5*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs_110);
    FAME_Matrix_Vector_Production_Qrs(y+6*size, temp_vec+6*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs_101);
    FAME_Matrix_Vector_Production_Qrs(y+7*size, temp_vec+7*size_temp, cuHandles, fft_buffer, Nx, Ny, Nz, Nd, D_kx, D_ky, D_kz, Pi_Qrs_011);

    hipFree( temp_vec );

}
